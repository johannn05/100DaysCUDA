#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define N 3  // matrix size

// function to perform matrix addition using cublas
void cublasMatrixAdd(float *h_A, float *h_B, float *h_C, int n) {
    float *d_A, *d_B, *d_C;
    
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    // initialize cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // perform C = A + B using cublasSaxpy (y = alpha * x + y)
    float alpha = 1.0f;
    hipblasSaxpy(handle, n * n, &alpha, d_A, 1, d_B, 1);

    hipMemcpy(h_C, d_B, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// main function
int main() {
    float h_A[N * N] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    float h_B[N * N] = {9, 8, 7, 6, 5, 4, 3, 2, 1};
    float h_C[N * N] = {0};

    cublasMatrixAdd(h_A, h_B, h_C, N);

    std::cout << "matrix C (A + B):" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_C[i * N + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
